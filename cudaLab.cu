#include<hip/hip_runtime.h>
 #include<cstdio>
 #include<iostream>
 
 using namespace std;
 
 __global__ void kernel(int* vector1,int n){
 	int idx = blockIdx.x*blockDim.x + threadIdx.x;
 		if(idx < n){
 			vector1[idx] *= 2;
	 	}
	return;
 }
 
 __host__ int main(){
 	int *vec1 = NULL;
 	int *cuvec1 = NULL;
 	hipStream_t stream1;
 	hipStream_t stream2;
 	hipStreamCreate(&stream1);
 	hipStreamCreate(&stream2);
 	int size = 1600;
 	int cusize = 160;
 	hipHostMalloc(&vec1,size * sizeof(int), hipHostMallocDefault);
 	for(int i = 0; i < size; i++){
 		vec1[i] = i;
	}
 	hipMalloc((void**)&cuvec1,cusize * sizeof(int));
 	int begin = 0;
 	int block = 32;
 	int grid = cusize/block;
 	while(begin < size){
		hipStreamCreate(&stream1);
        	hipStreamCreate(&stream2);
 		hipMemcpyAsync(cuvec1, vec1 + begin, cusize * sizeof(int), hipMemcpyHostToDevice,stream1);
 		kernel<<<grid,block,0,stream2>>>(cuvec1,cusize);
 		hipMemcpyAsync(vec1 + begin, cuvec1, cusize*sizeof(int), hipMemcpyDeviceToHost,stream1);
		hipDeviceSynchronize();
		hipStreamDestroy(stream1);
        	hipStreamDestroy(stream2);
 		begin += cusize;
	 }
	 for(int i = 0; i < size; i++){
	 	cout << vec1[i] << " ";
	 }
	 return 0;
 }

