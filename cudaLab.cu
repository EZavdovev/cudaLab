#include<hip/hip_runtime.h>
 #include<cstdio>
 #include<iostream>
 
 using namespace std;
 
 __global__ void kernel(int* vector1,int n){
 	int idx = blockIdx.x*blockDim.x + threadIdx.x;
 		if(idx < n){
 			vector1[idx] *= 2;
	 	}
	return;
 }
 
 __host__ int main(){
 	int *vec1 = NULL;
 	int *cuvec1 = NULL;
 	hipStream_t stream1;
 	hipStream_t stream2;
 	hipStreamCreate(&stream1);
 	hipStreamCreate(&stream2);
 	int size = 1600;
 	int cusize = 160;
 	hipHostMalloc(&vec1,size * sizeof(int), hipHostMallocDefault);
 	for(int i = 0; i < size; i++){
 		vec1[i] = i;
	}
 	hipMalloc((void**)&cuvec1, 2 * cusize * sizeof(int));
	hipMemcpy(cuvec1, vec1, cusize * sizeof(int), hipMemcpyHostToDevice);

 	int begin = cusize;
 	int block = 32;
	int useSeg = 1;
 	int grid = cusize/block;
 	while(begin < size){
		hipStreamCreate(&stream1);
        	hipStreamCreate(&stream2);
 		hipMemcpyAsync(cuvec1 + (useSeg) * cusize, vec1 + begin, cusize * sizeof(int), hipMemcpyHostToDevice,stream1);
 		kernel<<<grid,block,0,stream2>>>(cuvec1 + (1 - useSeg) * cusize,cusize);
 		hipMemcpyAsync(vec1 + (begin - cusize), cuvec1 + (1 - useSeg) * cusize, cusize*sizeof(int), hipMemcpyDeviceToHost,stream1);
		hipDeviceSynchronize();
		hipStreamDestroy(stream1);
        	hipStreamDestroy(stream2);
 		begin += cusize;
		useSeg = 1 - useSeg;
	 }
		
	kernel<<<grid,block>>>(cuvec1 + (1 - useSeg) * cusize,cusize);
	hipMemcpy(vec1 + (begin - cusize), cuvec1 + (1 - useSeg) * cusize, cusize*sizeof(int), hipMemcpyDeviceToHost);
	 for(int i = 0; i < size; i++){
	 	cout << vec1[i] << " ";
	 }
	 return 0;
 }

